#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <GL2/glew.h>
#include <GL2/freeglut.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <conio.h>
#include <time.h>

#define PI 3.14

struct Particle
{
	float x,y,vx,vy,m,r1 = rand()%255, r2 = rand() % 255, r3 = rand() % 255;
	Particle()
	{
		this->x = 0;
		this->y = 0;
		this->vx = 0;
		this->vy = 0;
		this->m = 0;
	}
	Particle(float x, float y, float vx, float vy, float m)
	{
		this->x = x;
		this->y = y;
		this->vx = vx;
		this->vy = vy;
		this->m = m;
	}
};

int N;

Particle *particles;

Particle* dev_part;

float floatRand() {

	return float(rand()) / (float(RAND_MAX) + 1.0);
}

__global__ void calculating(Particle* dev_part, int N)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	Particle &p0 = dev_part[i];
	for (int j = 0; j < N; ++j) {
		if (j == i)continue;
		const Particle &p = dev_part[j];
		float d = sqrt(pow((p0.x - p.x), 2) + pow((p0.y - p.y), 2));
		if (d > 3)
		{
			p0.vx += 0.00067 * p.m / pow(d, 2) * (p.x - p0.x) / d;
			p0.vy += 0.00067 * p.m / pow(d, 2) * (p.y - p0.y) / d;
		}
	}
	p0.x += p0.vx;
	p0.y += p0.vy;
}

void drawFilledCircle(GLfloat x, GLfloat y, GLfloat radius) {
	int triangleAmount = 20;
	GLfloat twicePi = 2.0f * PI;
	glBegin(GL_TRIANGLE_FAN);
	glVertex2f(x, y);
	for (int i = 0; i <= triangleAmount; i++) {
		glVertex2f(
			x + (radius * cos(i *  twicePi / triangleAmount)),
			y + (radius * sin(i * twicePi / triangleAmount))
		);
	}
	glEnd();
}


void display()
{
	glClear(GL_COLOR_BUFFER_BIT);
	//glBegin(GL_POINTS);
	for (int i = 0; i < N; ++i) {
		glColor3b(particles[i].r1, particles[i].r2, particles[i].r3);
		//glVertex2f(particles[i].x, particles[i].y);
		drawFilledCircle(particles[i].x, particles[i].y, 0.5);//particles[i].m
	}
	//glEnd();
	glutSwapBuffers();
}

void timer(int = 0)
{
	hipMemcpy(dev_part,particles,N*sizeof(Particle),hipMemcpyHostToDevice);
	calculating<<<1, N>>>(dev_part,N);
	hipDeviceSynchronize();
	hipMemcpy(particles, dev_part, N * sizeof(Particle), hipMemcpyDeviceToHost);
	display();
	glutTimerFunc(1, timer, 0);
}

int main(int argc, char **argv)
{
	bool circle = true;
	srand(time(0));
	while (circle) {
		system("cls");
		std::cout << "Welcome to the gravity modeling program! Choose your option to work with:\n1.Create objects with random parameters\n2.Create objects with custom parameteres\n3.Run default system \n0.Exit program\n";
		int key = _getch();
		switch (key)
		{
		case 48:
			circle = false;
			break;
		case 49:
			system("cls");
			std::cout << "Enter number of objects, but it should be in the interval [0,1024]:";
			do {
				std::cin >> N;
				if (N < 0|| N > 1024)std::cout << "Input is not correct. Please try again:";
			} while (N<0||N>1024);
			particles = (Particle*)malloc(N * sizeof(Particle));
			for (size_t i = 0; i < N; ++i)
			{
				particles[i] = Particle(rand() % 100 + 50, rand() % 100 + 50, floatRand() - 0.5, floatRand() - 0.5, rand() % 10);
			}
			std::cout << "Add heavy object to the system?(1-yes,0-no):";
			int ans;
			std::cin >> ans;
			switch (ans)
			{
			case 1:
				particles[0] = Particle(100, 100, 0, 0, 10000);
			default:
				break;
			}

			break;
		case 50:
			system("cls");
			std::cout << "Enter number of objects, but it should be in the interval [0,1024]:";
			do {
				std::cin >> N;
				if (N < 0 || N > 1024)std::cout << "Input is not correct. Please try again:";
			} while (N < 0 || N>1024);
			particles = (Particle*)malloc(N * sizeof(Particle));

			for (size_t i = 0; i < N; i++)
			{
				std::cout << "Entering parametrs of " << i+1 <<" particle\n";
				double x, y, vx, vy, m;
				std::cout << "Enter x coordinate:";
				do {
					std::cin >> x;
					if (x < 0 || x > 1024)std::cout << "Input is not correct. Please try again:";
				} while (x < 0);
				std::cout << "Enter y coordinate:";
				do {
					std::cin >> y;
					if (y < 0 || y > 1024)std::cout << "Input is not correct. Please try again:";
				} while (y < 0);
				std::cout << "Enter speed on x coordinate:";
				do {
					std::cin >> vx;
					if (vx < 0)std::cout << "Input is not correct. Please try again:";
				} while (vx < 0);
				std::cout << "Enter speed on y coordinate:";
				do {
					std::cin >> vy;
					if (vy < 0)std::cout << "Input is not correct. Please try again:";
				} while (vy < 0);
				std::cout << "Enter mass:";
				do {
					std::cin >> m;
					if (m < 0)std::cout << "Input is not correct. Please try again:";
				} while (m < 0);
				particles[i] = Particle(x, y, vx, vy, m);
			}
			break;
		case 51:
			N = 6;
			particles = (Particle*)malloc(N * sizeof(Particle));
			particles[0] = Particle(100, 100, 0, 0, 1000);//The Sun
			particles[1] = Particle(85, 100, 0, -0.23, 6);//Mercury
			particles[2] = Particle(40, 100, 0, 0.1, 10);//Earth
			particles[3] = Particle(25, 100, 0, -0.08, 5);//Mars
			particles[4] = Particle(70, 100, 0, 0.15, 8);//Venera
			particles[5] = Particle(35, 100, 0, 0.11, 0.1);//Moon
			break;
		default:
			circle = false;
			break;
		}
		if (circle) {
			hipMalloc((void**)&dev_part, N * sizeof(Particle));
			glutInit(&argc, argv);
			glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB);
			glutInitWindowSize(1000, 800);
			glutInitWindowPosition(400, 100);
			glutCreateWindow("Gravitation");
			glClearColor(0, 0, 0, 1.0);
			glMatrixMode(GL_PROJECTION);
			glLoadIdentity();
			glOrtho(0, 200, 200, 0, -1, 1);
			glutDisplayFunc(display);
			timer();
			glutMainLoop();
			free(particles);
			hipFree(dev_part);
		}
	}
}